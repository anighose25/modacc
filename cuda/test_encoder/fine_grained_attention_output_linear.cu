// %%cuda --name fine_grained.cu
#include "gemm.h"
#include "utils.h"


int main(int argc, char *argv[])
{
    // PLEASE NOTE: number of queues must be less than batch_size
    bool sync_flag = true;
    int batch_size = atoi(argv[1]);
    int sequence_length = atoi(argv[2]);
    int hidden_size = atoi(argv[3]);
    int intermediate_size = atoi(argv[4]);
    int nh = atoi(argv[5]);
    int nq = atoi(argv[6]);
    int bsz=batch_size*sequence_length;
    int bsz_seq=batch_size*sequence_length;
    /* if ( argc > 5 )
    {
        if ( std::string(argv[5]) == "sync" )
            sync_flag = true;
    } */

    std::array <int, 3> gemm_algos = {HIPBLAS_GEMM_DEFAULT, HIPBLAS_GEMM_DEFAULT, HIPBLAS_GEMM_DEFAULT};
    std::cout << "################################################################" << std::endl;
    std::cout << "batch size=" << batch_size << std::endl;
    std::cout << "sequence length=" << sequence_length << std::endl;
    std::cout << "hidden layer size=" << hidden_size << std::endl;
    std::cout << "sync flag=" << sync_flag << std::endl;
    std::cout << "################################################################" << std::endl;

    ScheduleEngine SE(30);
     Buffer<float> buf_1(batch_size * sequence_length * hidden_size, &SE);
    Buffer<float> attn_o_inp(batch_size * sequence_length * hidden_size, &SE);
    Buffer<float> attn_ow(batch_size * hidden_size * hidden_size, &SE);
    
FeedForward<float> _attn_out_linear(FeedForward<float>::Config(bsz, 
                                                              hidden_size, 
                                                              hidden_size,
                                                              gemm_algos,
                                                              true));
       Stopwatch sw;
    
    printf("\x1b[41;1mstarting profiling for fine grained implementation\x1b[0m\n");
    sw.restart();
    _attn_out_linear.ForwardCheckpointPartition(bsz_seq, &attn_o_inp, &attn_ow, &buf_1, &SE, nq,true);
    sw.stop();
    std::cout << "t(" << nq << ")=" << sw.GetTimeInSeconds() << std::endl;
    fileWrite("queue_size="+std::to_string(nq)+".txt", std::to_string(sw.GetTimeInSeconds()));

    return 0;
}