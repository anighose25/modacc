// %%cuda --name fine_grained.cu
#include "gemm.h"
#include "utils.h"


int main(int argc, char *argv[])
{
    // PLEASE NOTE: number of queues must be less than batch_size
    bool sync_flag = true;
    int batch_size = atoi(argv[1]);
    int sequence_length = atoi(argv[2]);
    int hidden_size = atoi(argv[3]);
    int intermediate_size = atoi(argv[4]);
    int nh = atoi(argv[5]);
    int nq = atoi(argv[6]);
    int bsz=batch_size*sequence_length;
    /* if ( argc > 5 )
    {
        if ( std::string(argv[5]) == "sync" )
            sync_flag = true;
    } */

    std::array <int, 3> gemm_algos = {HIPBLAS_GEMM_DEFAULT, HIPBLAS_GEMM_DEFAULT, HIPBLAS_GEMM_DEFAULT};
    std::cout << "################################################################" << std::endl;
    std::cout << "batch size=" << batch_size << std::endl;
    std::cout << "sequence length=" << sequence_length << std::endl;
    std::cout << "hidden layer size=" << hidden_size << std::endl;
    std::cout << "sync flag=" << sync_flag << std::endl;
    std::cout << "################################################################" << std::endl;

    ScheduleEngine SE(30);
    Buffer<float> ff1_inp(batch_size * sequence_length * hidden_size, &SE);
    Buffer<float> inter_w(hidden_size * intermediate_size, &SE);
    Buffer<float> ff2_inp(batch_size * sequence_length * intermediate_size, &SE);
 
    FeedForward<float> _ff1(FeedForward<float>::Config(bsz, 
                                                       intermediate_size,
                                                       hidden_size,
                                                       gemm_algos,
                                                       true));
    Stopwatch sw;
    
    printf("\x1b[41;1mstarting profiling for fine grained implementation\x1b[0m\n");
    sw.restart();
    _ff1.ForwardCheckpointPartition(batch_size*sequence_length, &ff1_inp, &inter_w, &ff2_inp, &SE, nq, true);
    sw.stop();
    std::cout << "t(" << nq << ")=" << sw.GetTimeInSeconds() << std::endl;
    fileWrite("queue_size="+std::to_string(nq)+".txt", std::to_string(sw.GetTimeInSeconds()));

    return 0;
}
