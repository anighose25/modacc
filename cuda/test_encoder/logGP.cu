#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <time.h>
#include "utils.h"
#include <stdio.h>
#include <assert.h>
#include <stdexcept>
#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>

#define DEV_QUERY 0
#define SEQ 1
#define ASYNC 1
#define max(a,b) (a>b)?a:b
#ifndef EVENT_PROFILE
#define EVENT_PROFILE 0
#endif
#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
    }                                                                          \
}

#define CHECK_CUBLAS(call)                                                     \
{                                                                              \
   hipblasStatus_t err;                                                        \
    if ((err = (call)) != HIPBLAS_STATUS_SUCCESS)                               \
    {                                                                          \
        fprintf(stderr, "Got CUBLAS error %d at %s:%d\n", err, __FILE__,       \
                __LINE__);                                                     \
        exit(1);                                                               \
    }                                                                          \
}


void initializeOnes(float *input, int num_elements)
{
    for(unsigned int i = 0; i < num_elements; i++)
    {
        input[i] = 1.0;
    }
}



float calculate_G(hipEvent_t start[], hipEvent_t stop[], int num_streams, long long int size)
{
    float event_recorded_time = 0, total_time =0;
    for(int i=0;i<num_streams;i++)
    {
       hipEventElapsedTime(&event_recorded_time, start[i], stop[i]);
       total_time += event_recorded_time; 
    }
    return total_time/size;
}

float calculate_g(hipEvent_t start[], hipEvent_t stop[], int num_streams, long long int size)
{
    float event_recorded_time = 0, total_time =0;
    for(int i=1;i<num_streams;i++)
    {
       hipEventElapsedTime(&event_recorded_time, start[i],stop[i-1]);
       //printf("Event recorded time for g : %f\n",event_recorded_time);
       total_time += event_recorded_time; 
    }
    return total_time/num_streams;
}



int main(int argc, char *argv[])
{
    //Query Device
#if DEV_QUERY
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("> %s Starting...\n", argv[0]);
    printf("> Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // check if device supports hyper-q
    if (deviceProp.major < 3 || (deviceProp.major == 3 && deviceProp.minor < 5))
    {
        if (deviceProp.concurrentKernels == 0)
        {
            printf("> GPU does not support concurrent kernel execution (SM 3.5 "
                    "or higher required)\n");
            printf("> CUDA kernel runs will be serialized\n");
        }
        else
        {
            printf("> GPU does not support HyperQ\n");
            printf("> CUDA kernel runs will have limited concurrency\n");
        }
    }

    printf("> Compute Capability %d.%d hardware with %d multi-processors\n",
            deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount);

#endif
    //Variable Initialization
    
    
    
    
    long long int size = atoi(argv[1]);
    int num_streams = atoi(argv[2]);
    


    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));
    
    size_t nbytes_A = size*sizeof(float);
   
    Stopwatch sw, tsync;


    // host array creation (page-locked)

    float *h_input = NULL; 
    CHECK(hipHostMalloc((void **)&h_input, nbytes_A));
    initializeOnes(h_input, size);

    
    // device memory creation

    float *d_A = NULL;

    CHECK(hipMalloc((void **)&d_A, nbytes_A));


    // stream configuration


    hipStream_t memory[num_streams];

    hipEvent_t start[num_streams], stop[num_streams];
    hipEvent_t seq_start, seq_end;
    CHECK(hipEventCreate(&seq_start));
    CHECK(hipEventCreate(&seq_end));
    for(int i=0;i<num_streams;i++)
    {
        CHECK(hipEventCreate(&start[i]));
        CHECK(hipEventCreate(&stop[i]));
    }


    for(int i=0;i<num_streams;i++)
    {
        CHECK(hipStreamCreate(&memory[i]));
    }



#if SEQ

    sw.start();

    //H2D Copy of 1 byte

    hipEventRecord(seq_start);
    CHECK(hipMemcpy(d_A, h_input, 1, hipMemcpyHostToDevice));
    hipEventRecord(seq_end);
    float event_recorded_time = 0;
    CHECK(hipDeviceSynchronize());    
    hipEventElapsedTime(&event_recorded_time, seq_start, seq_end);
    printf("L+o for H2D: %lf\n",event_recorded_time);

    hipEventRecord(seq_start);
    CHECK(hipMemcpy(h_input, d_A, 1, hipMemcpyDeviceToHost));
    hipEventRecord(seq_end);
    CHECK(hipDeviceSynchronize());    
    hipEventElapsedTime(&event_recorded_time, seq_start, seq_end);
    printf("L+o for D2H: %lf\n",event_recorded_time);


#endif


#if ASYNC

// Multistream implementation 

    int granularity = num_streams;
    int offset = 0;
    int buffer_offset_A;
    int sub_A =  size/granularity;    
    size_t sub_nbytes_A = nbytes_A/granularity;
    sw.start();
    for(int i=0;i<num_streams;i++)
    {
        buffer_offset_A = i*sub_A;    
        hipEventRecord(start[i],memory[i]);
        CHECK(hipMemcpyAsync(&d_A[buffer_offset_A], &h_input[buffer_offset_A], sub_nbytes_A,
                              hipMemcpyHostToDevice, memory[i]));
        hipEventRecord(stop[i],memory[i]);
    }
    CHECK(hipDeviceSynchronize());    
    sw.stop();

    printf("H2D Time:%f\n", sw.GetTimeInSeconds());
    for(int i=0;i<num_streams;i++)
    {
        hipEventSynchronize(stop[i]);
    }    
    printf("G for H2D: %f\n",calculate_G(start,stop,num_streams,size));
    printf("g for H2D: %f\n",calculate_g(start,stop,num_streams,size));
    buffer_offset_A=0;
    sw.restart();
    for(int i=0;i<num_streams;i++)
    {
        buffer_offset_A = i*sub_A;    
        hipEventRecord(start[i],memory[i]);
        CHECK(hipMemcpyAsync(&h_input[buffer_offset_A], &d_A[buffer_offset_A], sub_nbytes_A,
                              hipMemcpyDeviceToHost, memory[i]));
        hipEventRecord(stop[i],memory[i]);
    }

    CHECK(hipDeviceSynchronize());   
    sw.stop(); 
    printf("D2H Time:%f\n", sw.GetTimeInSeconds());
    for(int i=0;i<num_streams;i++)
    {
        hipEventSynchronize(stop[i]);
    }    
    printf("G for D2H: %f\n",calculate_G(start,stop,num_streams,size)); 
    printf("g for D2H: %f\n",calculate_g(start,stop,num_streams,size));
#endif
   
}
